#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <dirent.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#pragma pack(push, 1)
typedef struct {
    uint16_t type;              // Magic identifier: 0x4d42
    uint32_t size;              // File size in bytes
    uint16_t reserved1;         // Not used
    uint16_t reserved2;         // Not used
    uint32_t offset;            // Offset to image data in bytes
} BMPHeader;

typedef struct {
    uint32_t size;              // Header size in bytes
    int32_t width;              // Width of the image
    int32_t height;             // Height of the image
    uint16_t planes;            // Number of color planes
    uint16_t bits;              // Bits per pixel
    uint32_t compression;       // Compression type
    uint32_t imagesize;         // Image size in bytes
    int32_t xresolution;        // Pixels per meter
    int32_t yresolution;        // Pixels per meter
    uint32_t ncolors;           // Number of colors
    uint32_t importantcolors;   // Important colors
} BMPInfoHeader;
#pragma pack(pop)

typedef struct {
    unsigned char r, g, b;
} Pixel;

__global__ void process_frame_kernel(unsigned char** d_raw_data, int width, int height, int padding, int index) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int rawIdx = y * (width * 3 + padding) + x * 3;
        int pixelIdx = y * width + x;

        d_raw_data[index][rawIdx] = 255 - d_raw_data[index][rawIdx];
        d_raw_data[index][rawIdx + 1] = 255 - d_raw_data[index][rawIdx + 1];
        d_raw_data[index][rawIdx + 2] = 255 - d_raw_data[index][rawIdx + 2];
    }
}

// Function to read BMP file and decode pixel data
void read_bmp(const char* filename, unsigned char** raw_data, int* width, int* height) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        fprintf(stderr, "Could not open file: %s\n", filename);
        *raw_data = NULL;
        return;
    }

    BMPHeader header;
    BMPInfoHeader info;

    // Read BMP header and info
    fread(&header, sizeof(BMPHeader), 1, file);
    fread(&info, sizeof(BMPInfoHeader), 1, file);

    // Verify that it's a BMP file
    if (header.type != 0x4D42) {
        fprintf(stderr, "Not a BMP file\n");
        fclose(file);
        return;
    }

    // Verify color depth
    if (info.bits != 24) {
        fprintf(stderr, "Only 24-bit BMP files are supported\n");
        fclose(file);
        *raw_data = NULL;
        return;
    }

    *width = info.width;
    *height = info.height;

    int padding = (4 - (info.width * 3) % 4) % 4;
    int rawSize = (*height) * ((*width) * 3 + padding);

    // Allocate memory for raw data
    *raw_data = (unsigned char*)malloc(rawSize);

    // Read raw pixel data
    fseek(file, header.offset, SEEK_SET);
    fread(*raw_data, rawSize, 1, file);
    fclose(file);
}

void save_bmp(const char* filename, unsigned char* raw_data, int width, int height) {
    FILE* file = fopen(filename, "wb");
    if (!file) {
        fprintf(stderr, "Could not create file: %s\n", filename);
        return;
    }

    int padding = (4 - (width * 3) % 4) % 4;
    int rawSize = height * (width * 3 + padding);

    BMPHeader header = {
        .type = 0x4D42,
        .size = sizeof(BMPHeader) + sizeof(BMPInfoHeader) + rawSize,
        .reserved1 = 0,
        .reserved2 = 0,
        .offset = sizeof(BMPHeader) + sizeof(BMPInfoHeader)
    };

    BMPInfoHeader info = {
        .size = sizeof(BMPInfoHeader),
        .width = width,
        .height = height,
        .planes = 1,
        .bits = 24,
        .compression = 0,
        .imagesize = rawSize,
        .xresolution = 2835,  // 72 DPI
        .yresolution = 2835,  // 72 DPI
        .ncolors = 0,
        .importantcolors = 0
    };


    // Write BMP header and info
    fwrite(&header, sizeof(BMPHeader), 1, file);
    fwrite(&info, sizeof(BMPInfoHeader), 1, file);

    // Write raw pixel data
    fwrite(raw_data, rawSize, 1, file);
    fclose(file);
    free(raw_data);
}

void run_ffmpeg_command(const char* command) {
    int ret = system(command);
    if (ret != 0) {
        fprintf(stderr, "Error running command: %s\n", command);
        exit(1);
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <input.mp4> <output.mp4>\n", argv[0]);
        return 1;
    }

    const char* input_video = argv[1];
    const char* output_video = argv[2];

    // Create frames directory
    mkdir("frames", 0777);
    mkdir("output", 0777);

    // Extract frames from video
    char ffmpeg_command[256];
    snprintf(ffmpeg_command, sizeof(ffmpeg_command), "ffmpeg -i %s frames/frame_%%04d.bmp", input_video);
    run_ffmpeg_command(ffmpeg_command);

    auto start = std::chrono::high_resolution_clock::now();

    // Process frames
    DIR* dir;
    struct dirent* entry;
    if (!(dir = opendir("frames"))) {
        fprintf(stderr, "Could not open frames directory\n");
        return 1;
    }

    int num_frames = 0;
    while ((entry = readdir(dir)) != NULL) {
        if (strstr(entry->d_name, ".bmp") != NULL) {
            num_frames++;
        }
    }
    closedir(dir);

    unsigned char** raw_data = (unsigned char**)malloc(num_frames * sizeof(unsigned char*));
    int* widths = (int*)malloc(num_frames * sizeof(int));
    int* heights = (int*)malloc(num_frames * sizeof(int));

    for (int i = 0; i < num_frames; ++i) {
        char frame_filename[256];
        snprintf(frame_filename, sizeof(frame_filename), "frames/frame_%04d.bmp", i + 1);
        read_bmp(frame_filename, &raw_data[i], &widths[i], &heights[i]);
        // printf("Frame %d: %d x %d\n", i, widths[i], heights[i]);
        if (raw_data[i] == NULL) {
            fprintf(stderr, "Failed to read BMP file: %s\n", frame_filename);
            return 1;
        }
    }

    unsigned char** d_raw_data_array;
    hipHostMalloc((void***)&d_raw_data_array, num_frames * sizeof(unsigned char*), hipHostMallocDefault);

    // It is assumed that all frames have the same dimensions
    int padding = ((4 - (widths[0] * 3) % 4) % 4);
    int rawSize = heights[0] * (widths[0] * 3 + padding);

    for (int i = 0; i < num_frames; ++i) {
        hipMalloc((void**)&d_raw_data_array[i], rawSize);
    }

    int num_streams = 8;
    hipStream_t streams[num_streams];

    for (int i = 0; i < num_streams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    
    dim3 blockSize(16, 16);
    dim3 gridSize((widths[0] + blockSize.x - 1) / blockSize.x, 
                    (heights[0] + blockSize.y - 1) / blockSize.y);
    for (int i = 0; i < num_frames; ++i) {
        hipMemcpyAsync(d_raw_data_array[i], raw_data[i], rawSize, hipMemcpyHostToDevice, streams[i % num_streams]);
        process_frame_kernel<<<gridSize, blockSize, 0, streams[i % num_streams]>>>(d_raw_data_array, widths[i], heights[i], padding, i);
        hipMemcpyAsync(raw_data[i], d_raw_data_array[i], widths[i] * heights[i] * sizeof(Pixel), hipMemcpyDeviceToHost, streams[i % num_streams]);
    }

    hipDeviceSynchronize();

    for (int i = 0; i < num_frames; ++i) {
        hipFree(d_raw_data_array[i]);
    }

    hipFree(d_raw_data_array);

    for (int i = 0; i < num_streams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    for (int i = 0; i < num_frames; ++i) {
        char output_filename[256];
        snprintf(output_filename, sizeof(output_filename), "output/frame_%04d.bmp", i + 1);
        save_bmp(output_filename, raw_data[i], widths[i], heights[i]);
    }

    free(raw_data);
    free(widths);
    free(heights);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    printf("Total runtime: %f seconds\n", duration.count());

    // Combine frames into video
    snprintf(ffmpeg_command, sizeof(ffmpeg_command), "ffmpeg -framerate 30 -i output/frame_%%04d.bmp %s", output_video);
    run_ffmpeg_command(ffmpeg_command);
    
    // Remove frames directory
    snprintf(ffmpeg_command, sizeof(ffmpeg_command), "rm -rf frames");
    run_ffmpeg_command(ffmpeg_command);

    // Remove output directory
    snprintf(ffmpeg_command, sizeof(ffmpeg_command), "rm -rf output");
    run_ffmpeg_command(ffmpeg_command);
    


    return 0;
}